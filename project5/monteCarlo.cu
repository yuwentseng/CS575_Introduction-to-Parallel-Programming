#include "hip/hip_runtime.h"
// Array multiplication: C = A * B:

// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"


#ifndef BLOCKSIZE
#define BLOCKSIZE		128		// number of threads per block
#endif

#ifndef NUMTRIALS
#define NUMTRIALS			1024*1024 // array size
#endif

#ifndef TOLERANCE
#define TOLERANCE		0.00001f	// tolerance to relative error
#endif

// ranges for the random numbers:
const float XCMIN =	 0.0;
const float XCMAX =	 2.0;
const float YCMIN =	 0.0;
const float YCMAX =	 2.0;
const float RMIN  =	 0.5;
const float RMAX  =	 2.0;

// function prototypes:
float		Ranf( float, float );
int			Ranf( int, int );
void		TimeOfDaySeed( );

// monte carlo (CUDA Kernel) on the device
__global__  void MonteCarlo( float *xcs, float *ycs, float *rs, int *numHits )
{
    __shared__ int prods[BLOCKSIZE];

	unsigned int numItems = blockDim.x;
	unsigned int tnum = threadIdx.x;
	unsigned int wgNum = blockIdx.x;
	unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;

    float xc = xcs[gid];
    float yc = ycs[gid];
    float  r =  rs[gid];

    // solve for the intersection using the quadratic formula:
    float a = 2.;
    float b = -2.*( xc + yc );
    float c = xc*xc + yc*yc - r*r;
    float d = b*b - 4.*a*c;

    if( d < 0. )
    {
        prods[tnum] = 0;
    }
    else
    {
        // hits the circle:
        // get the first intersection:
        d = sqrt( d );
        float t1 = (-b + d ) / ( 2.*a );	// time to intersect the circle
        float t2 = (-b - d ) / ( 2.*a );	// time to intersect the circle
        float tmin = t1 < t2 ? t1 : t2;		// only care about the first intersection

        if( tmin < 0. )
        {
            prods[tnum] = 0;
        }
        else
        {
            // where does it intersect the circle?
            float xcir = tmin;
            float ycir = tmin;

            // get the unitized normal vector at the point of intersection:
            float nx = xcir - xc;
            float ny = ycir - yc;
            float n = sqrt( nx*nx + ny*ny );
            nx /= n;	// unit vector
            ny /= n;	// unit vector

            // get the unitized incoming vector:
            float inx = xcir - 0.;
            float iny = ycir - 0.;
            float in = sqrt( inx*inx + iny*iny );
            inx /= in;	// unit vector
            iny /= in;	// unit vector

            // get the outgoing (bounced) vector:
            float dot = inx*nx + iny*ny;
            //float outx = inx - 2.*nx*dot;	// angle of reflection = angle of incidence`
            float outy = iny - 2.*ny*dot;	// angle of reflection = angle of incidence`

            // find out if it hits the infinite plate:
            float t = ( 0. - ycir ) / outy;

            if( t < 0. )
            {
                prods[tnum] = 0;
            }
            else
            {
                prods[tnum] = 1;
            }
        }
    }

	for (int offset = 1; offset < numItems; offset *= 2)
	{
		int mask = 2 * offset - 1;
		__syncthreads();
		if ((tnum & mask) == 0)
		{
			prods[tnum] += prods[tnum + offset];
		}
	}

	__syncthreads();
	if (tnum == 0)
		numHits[wgNum] = prods[0];
}

// helper functions
float Ranf( float low, float high )
{
        float r = (float) rand();               // 0 - RAND_MAX
        float t = r  /  (float) RAND_MAX;       // 0. - 1.

        return   low  +  t * ( high - low );
}

int Ranf( int ilow, int ihigh )
{
        float low = (float)ilow;
        float high = ceil( (float)ihigh );

        return (int) Ranf(low,high);
}

void TimeOfDaySeed( )
{
	struct tm y2k = { 0 };
	y2k.tm_hour = 0;   y2k.tm_min = 0; y2k.tm_sec = 0;
	y2k.tm_year = 100; y2k.tm_mon = 0; y2k.tm_mday = 1;

	time_t  timer;
	time( &timer );
	double seconds = difftime( timer, mktime(&y2k) );
	unsigned int seed = (unsigned int)( 1000.*seconds );    // milliseconds
	srand( seed );
}

// main program:

int
main( int argc, char* argv[ ] )
{
	int dev = findCudaDevice(argc, (const char **)argv);

	// allocate host memory:
    
	float * hxcs = new float [ NUMTRIALS ];
	float * hycs = new float [ NUMTRIALS ];
	float * hrs = new float [ NUMTRIALS ];
    int * hnumHits = new int [ NUMTRIALS/BLOCKSIZE ];
    
    // fill the random-value arrays:
    for( int n = 0; n < NUMTRIALS; n++ )
    {
        hxcs[n] = Ranf( XCMIN, XCMAX );
        hycs[n] = Ranf( YCMIN, YCMAX );
        hrs[n] = Ranf(  RMIN,  RMAX );
    }
    
	// allocate device memory:

    float *dxcs, *dycs, *drs;
    int *dnumHits;
    
	dim3 dimsxcs( NUMTRIALS, 1, 1 );
	dim3 dimsycs( NUMTRIALS, 1, 1 );
	dim3 dimsrs( NUMTRIALS, 1, 1 );
    dim3 dimsnumHits( NUMTRIALS, 1, 1);
    

	hipError_t status;

	status = hipMalloc( reinterpret_cast<void **>(&dxcs), NUMTRIALS*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dycs), NUMTRIALS*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&drs), NUMTRIALS*sizeof(float) );
		checkCudaErrors( status );
    status = hipMalloc( reinterpret_cast<void **>(&dnumHits), (NUMTRIALS/BLOCKSIZE)*sizeof(int) );
		checkCudaErrors( status );


	// copy host memory to the device:

	status = hipMemcpy( dxcs, hxcs, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( dycs, hycs, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
    status = hipMemcpy( drs, hrs, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );

	// setup the execution parameters:

	dim3 threads(BLOCKSIZE, 1, 1 );
	dim3 grid( NUMTRIALS / threads.x, 1, 1 );

	// Create and start timer

	hipDeviceSynchronize( );

	// allocate CUDA events that we'll use for timing:

	hipEvent_t start, stop;
	status = hipEventCreate( &start );
		checkCudaErrors( status );
	status = hipEventCreate( &stop );
		checkCudaErrors( status );

	// record the start event:

	status = hipEventRecord( start, NULL );
		checkCudaErrors( status );

	// execute the kernel:

    MonteCarlo<<< grid, threads >>>( dxcs, dycs, drs, dnumHits );

	// record the stop event:

	status = hipEventRecord( stop, NULL );
		checkCudaErrors( status );

	// wait for the stop event to complete:

	status = hipEventSynchronize( stop );
		checkCudaErrors( status );

	float msecTotal = 0.0f;
	status = hipEventElapsedTime( &msecTotal, start, stop );
		checkCudaErrors( status );

	// compute and print the performance

	double secondsTotal = 0.001 * (double)msecTotal;
	double trialsPerSecond = (float)NUMTRIALS / secondsTotal;
	double megaTrialsPerSecond = trialsPerSecond / 1000000.;
	fprintf( stderr, "NUMTRIALS =%10d, MegaTrials/Second =%10.2lf\n", NUMTRIALS, megaTrialsPerSecond );

	// copy result from the device to the host:

	status = hipMemcpy( hnumHits, dnumHits, (NUMTRIALS/BLOCKSIZE)*sizeof(int), hipMemcpyDeviceToHost );
		checkCudaErrors( status );
    
    int sum = 0.;
	for(int i = 0; i < NUMTRIALS/BLOCKSIZE; i++ )
	{
		sum += hnumHits[i];
	}
    float prob = (float)(sum) / (float)(NUMTRIALS);
    fprintf( stderr, "\nNumHit =%10d\n", sum );
	fprintf( stderr, "\nProbability =%8.4lf\n", prob );
    
	// clean up memory:
    delete [ ] hxcs;
    delete [ ] hycs;
    delete [ ] hrs;
    delete [ ] hnumHits;

    status = hipFree( dxcs );
		checkCudaErrors( status );
	status = hipFree( dycs );
		checkCudaErrors( status );
	status = hipFree( drs );
		checkCudaErrors( status );
    status = hipFree( dnumHits );
		checkCudaErrors( status );

	return 0;
}